#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int modprodCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExpCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodCuda(aExpb, z, p);
    z = modprodCuda(z, z, p);
    b /= 2;
  }
  return aExpb;
}



//device function
__global__ void kernelDecrypt(int n, int p, int g, int h, int x) {

  int threadid = threadIdx.x; //thread number
  int Nblock = blockDim.x;  //number of threads in a block
	int blockid = blockIdx.x; //block number

  int id = threadid + blockid*Nblock;


	if (x==0 || modExpCuda(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExpCuda(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
}




int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
	int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
	char buf[bufferSize];
  FILE * fp;
  fp = fopen("public_key.txt", "r");
  if(!fp) {
    return 1;
  }

	fgets(buf, bufferSize, fp);
	sscanf(buf, "%d", &n);
	fgets(buf, bufferSize, fp);
	sscanf(buf, "%d", &p);
	fgets(buf, bufferSize, fp);
	sscanf(buf, "%d", &g);
	fgets(buf, bufferSize, fp);
	sscanf(buf, "%d", &h);

	fclose(fp);

	printf("%d, %d, %d, %d\n", n, p, g, h);
	fp = fopen("message.txt", "r");
	fgets(buf, bufferSize, fp);
	sscanf(buf, "%u", &Nints);

	unsigned int charsPerInt = (n-1) / 8;
  unsigned int Nchars = charsPerInt*Nints;

  //storage for message as elements of Z_p
  unsigned int *z = (unsigned int *) malloc(Nints*sizeof(unsigned int)); 
  
  //storage for extra encryption coefficient 
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
	
	int i;
	for(i = 0; i < Nints; i++) {
		fgets(buf,bufferSize, fp);
		sscanf(buf, "%u%u", &z[i], &a[i]);
		printf("Z[i]: %u, A[i]: %u\n", z[i], a[i]);
	}

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */
  //Decrypt the Zmessage with the ElGamal cyrptographic system
  ElGamalDecrypt(z,a,Nints,p,x);

  convertZToString(z, Nints, message, Nchars);

  printf("Decrypted Message = \"%s\"\n", message);
  printf("\n");

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
	//Device array
  double *d_x;

  //allocate memory on the Device with hipMalloc
  hipMalloc(&d_x, sizeof(double));


	int Nthreads = atoi(argv[2]); //get the number of threads per block from command line
  int Nblocks = (N+Nthreads-1)/Nthreads;


  kernelDecrypt <<<Nblocks ,Nthreads >>>(d_x);

	//copy data from the device to the host
  hipMemcpy(x, d_x, sizeof(double), hipMemcpyDeviceToHost);


  return 0;
}
